#include "hip/hip_runtime.h"
#include"hip/hip_runtime.h"
#include<iostream>
#include<stdlib.h>
using namespace std;

__global__ void matrixMul(float *a, float *b, float *c, int m,int n,int k)
{
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int column = blockIdx.x*blockDim.x+threadIdx.x;
    int sum=0;
    for(int i=0;i<n;i++)
        sum+=a[row*n+i]*b[i*k+column];
    c[row*k+column]=sum;
}

void multiply(float *h_subA,float *h_subB,float *h_subC,int subA_m, int subA_n,int subB_k, int id)
{
    float *d_subA, *d_subB, *d_subC;
    hipSetDevice(id);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp,id);
    cout<<"Device["<<id<<"]"<<deviceProp.name<<endl;
    hipMalloc(&d_subA,subA_m*subA_n*sizeof(float));
    hipMalloc(&d_subB,subA_n*subB_k*sizeof(float));
    hipMalloc(&d_subC,subA_m*subB_k*sizeof(float));

    hipMemcpy(d_subA,h_subA,subA_m*subA_n*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_subB,h_subB,subA_n*subB_k*sizeof(float),hipMemcpyHostToDevice);

    int threads_per_block = 16;
    dim3 block_size(threads_per_block,threads_per_block);
    dim3 grid_size(subB_k/block_size.x,subA_m/block_size.y);
    matrixMul<<<grid_size,block_size>>>(d_subA,d_subB,d_subC,int subA_m, int subA_n,int subB_k);
    hipMemcpy(h_subC,d_subC,subA_m*subB_k*sizeof(float),hipMemcpyDeviceToHost);
    hipFree(d_subA);
    hipFree(d_subB);
    hipFree(d_subC);
}